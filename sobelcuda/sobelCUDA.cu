#include "hip/hip_runtime.h"
#include <opencv2/core.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/core/cuda.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <math.h>
#include <iostream>

#include <sys/time.h>

// #include <pthread.h>

// #include <omp.h>

using namespace cv;

struct argsSobel
{
   Mat *img;
   Mat *out;
   int idxI;
   int idxF;
   int th;
};

struct argsBlur
{
   Mat *img;
   Mat *out;
   int idxI;
   int idxF;
   int blurSize;
   double **kernel;
};

std::string name;

void gaussKernel(int size, double sigma, int K, double * out)
{
   double sum = 0;
   int i, j;

   for (int i = 0; i < size; i++)
   {
      for (j = 0; j < size; j++)
      {
         double x = i - (size - 1) / 2.0;
         double y = j - (size - 1) / 2.0;
         out[i * size + j] = K * exp(((pow(x, 2) + pow(y, 2)) / ((2 * pow(sigma, 2)))) * (-1));
         sum += out[i * size + j];
      }
   }

   for (i = 0; i < size; i++)
      for (j = 0; j < size; j++)
         out[i * size + j] /= sum;

}

__global__ void gaussBlur(double* kernel, int blurSize, uchar* img, int img_step, int img_cols, int img_rows, uchar* out)
{
   int index = blockIdx.x * blockDim.x + threadIdx.x;
   int stride = blockDim.x * gridDim.x;

   for (int idx = index; idx < img_cols * img_rows; idx += stride)
   {
      int i = idx % img_cols;
      int j = idx / img_cols;

      float sum = 0.0;

      for (int k = 0; k < blurSize; k++)
      {
         for (int l = 0; l < blurSize; l++)
         {
            int x = i + (k - blurSize / 2);
            int y = j + (l - blurSize / 2);

            if (x >= 0 && x < img_cols && y >= 0 && y < img_rows)
            {
               double kV = kernel[k * blurSize + l];
               sum += img[y * img_cols + x] * kV;
            }
         }
      }

      out[idx] = sum;
   }
}

__global__ void sobel(int treshold, uchar* img, int img_step, int img_cols, int img_rows, uchar* out)
{

   int kS = 3;
   double sobelX[3][3] = {
       {1.0, 0.0, -1.0},
       {2.0, 0.0, -2.0},
       {1.0, 0.0, -1.0}};

   double sobelY[3][3] = {
       {1.0, 2.0, 1.0},
       {0.0, 0.0, 0.0},
       {-1.0, -2.0, -1.0}};

   int index = blockIdx.x * blockDim.x + threadIdx.x;
   int stride = blockDim.x * gridDim.x;

   for (int idx = index; idx < img_cols * img_rows; idx += stride)
   {
      int i = idx % img_cols;
      int j = idx / img_cols;

      float sumX = 0.0;
      float sumY = 0.0;

      for (int k = 0; k < kS; k++)
      {
         for (int l = 0; l < kS; l++)
         {
            int x = i + (k - kS / 2);
            int y = j + (l - kS / 2);

            if (x >= 0 && x < img_cols && y >= 0 && y < img_rows)
            {
               sumX += sobelX[k][l] * img[y * img_cols + x];
               sumY += sobelY[k][l] * img[y * img_cols + x];
            }
         }
      }

      int v = (int)(sqrt(sumX * sumX + sumY * sumY) / 1448 * 256);
      int a = max(v, treshold);

      if (a == treshold) out[idx] = 0;
      else out[idx] = 255;
   }
}

void bW(Mat *img)
{
   int rows = ((Mat)*img).rows;
   int cols = ((Mat)*img).cols;

   for (int i = 0; i < rows; i++)
   {
      for (int j = 0; j < cols; j++)
      {
         Vec3b p = ((Mat)*img).at<Vec3b>(i, j);
         unsigned char gray = p[0] * 0.3 + p[1] * 0.58 + p[2] * 0.11;
         ((Mat)*img).at<Vec3b>(i, j) = Vec3b(gray, gray, gray);
      }
   }
}

float sobelSq(std::string iName, int th, bool blr, int bS, int bSg, int blockSize, int blocks)
{
   name = iName;

   struct timeval tval_before, tval_after, tval_result;
   gettimeofday(&tval_before, NULL);

   std::string image_path = samples::findFile("images/" + name);
   Mat imgFC = imread(image_path, IMREAD_COLOR);
   Mat img;

   cvtColor(imgFC, img, cv::COLOR_BGRA2GRAY);

   
   uchar* p_img;
   uchar* p_out;
   hipMallocManaged(&p_img, sizeof(uchar) * img.rows * img.cols);
   hipMallocManaged(&p_out, sizeof(uchar) * img.rows * img.cols);

   Mat in(img.rows, img.cols, img.type(), p_img);
   Mat out(img.rows, img.cols, img.type(), p_out);

   cuda::GpuMat gpuImg(img.rows, img.cols, img.type(), p_img);
   cuda::GpuMat gpuOut(img.rows, img.cols, img.type(), p_out);

   img.copyTo(in);
   gpuImg.upload(img);

   img.release();

   double *kernel;
   hipMallocManaged(&kernel, sizeof(double) * bS * bS );

   // ----------------- BLUR-----------------
   if (blr)
   {
      gaussKernel(bS, bSg, 1, kernel);     

      gaussBlur<<<blocks, blockSize>>>((double *) kernel, bS, (uchar *) p_img, gpuImg.step, gpuImg.cols, gpuImg.rows, (uchar *) p_out);
      hipDeviceSynchronize();

      hipError_t error = hipGetLastError();
      if (error != hipSuccess) {
         fprintf(stderr, "ERROR: %s \n", hipGetErrorString(error));
      }
   }

   out.copyTo(in);
   
   // ----------------- SOBEL-----------------

   sobel<<<blocks, blockSize>>>(th, (uchar *) p_img, gpuImg.step, gpuImg.cols, gpuImg.rows, (uchar *) p_out);
   hipDeviceSynchronize();

   hipError_t error = hipGetLastError();
   if (error != hipSuccess) {
      fprintf(stderr, "ERROR: %s \n", hipGetErrorString(error));
   }

   gettimeofday(&tval_after, NULL);
   timersub(&tval_after, &tval_before, &tval_result);
   // printf("%d threads -> %ld.%06lds\n", p, (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);

   imwrite("out/" + name, out);

   // free(prms);
   out.release();
   in.release();

   gpuImg.release();
   gpuOut.release();

   hipFree(p_img);
   hipFree(p_out);
   hipFree(kernel);

   // String res = tval_result.tv_sec + "." + tval_result.tv_usec;
   // std::cout << res << std::endl;
   return (long int)tval_result.tv_sec + (long int)tval_result.tv_usec * 0.000001;
}