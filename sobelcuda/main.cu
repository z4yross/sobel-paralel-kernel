#include "hip/hip_runtime.h"
// make
// ./main -i t_640p.jpg -T 7 -B 10 -b 1 -k 7 -t 2 -m 1 -u 10

#include "sobelCUDA.cu"
#include <unistd.h>


void usage(const char *argv0){
   fprintf(stderr, "Usage: %s [-i nombreImagen][-T hilos][-B bloques][-t sobelTreshold][-b blur][-k blurSize][-s blurSigma][-m calcularPromedio][-u intentos][-h help-]\n", argv0);
   exit(EXIT_FAILURE);
}

int main(int argc, char **argv){
   std::string name = "gears.jpg";
   int treshold = 0;

   bool blur = true;
   int blurSize = 3;
   int blurSigma = 100;
   int blockSize = 1;
   bool dbg = false;
   int meanTr = 10;
   int blocks = 1;

   int opt;

   while ((opt = getopt(argc, argv, "i:T:B:t:b:k:s:m:u:h")) != -1){
      switch (opt){
         case 'i':
            name = optarg;
            break;
         case 'T':
            blockSize = atoi(optarg);
            break;
         case 'B':
            blocks = atoi(optarg);
            break;
         case 't':
            treshold = atoi(optarg);
            break;
         case 'b':
            blur = atoi(optarg) == 0 ? false : true;
            break;
         case 'k':
            blurSize = atoi(optarg);
            break;
         case 's':
            blurSigma = atoi(optarg);
            break;
         case 'm':
            dbg = atoi(optarg) >= 0 ? true : false;
            break;
         case 'u':
            meanTr = atoi(optarg);
            break;
         case 'h':
            usage(argv[0]);
         default:
            usage(argv[0]);
      }   
   }

   if(dbg){   
      for(int i = 0; i <= blockSize; i++){
         int p = pow(2, i);

         double sum = 0.0;

         for(int j = 0; j < meanTr; j++){
            sum += sobelSq(name, treshold, blur, blurSize, blurSigma, p, blocks);
         }

         sum /= meanTr;

         printf("%d threads %d blocks -> %fs mean\n", p, blocks, sum);
      }
   }else{
      float res = sobelSq(name, treshold, blur, blurSize, blurSigma, blockSize, blocks);
      printf("%d threads %d blocks -> %fs\n", blockSize, blocks, res);
   }


   return 0;
}
